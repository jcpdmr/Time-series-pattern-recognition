#include "hip/hip_runtime.h"
#include "cuda_utility.h"

int main() {
    FILTER_TYPE flt_type = ZMNCC;

    // Open the file
    ifstream file("../input_data/household_power_consumption.txt");
    if (!file.is_open()) {
        cerr << "Error opening the file!" << endl;
        return 1;
    }

    vector<string> dates;
    vector<float> values;
	dates.reserve(SERIES_LENGTH);
	values.reserve(SERIES_LENGTH);

    string line;

	// Skip first line (because it contains the header row)
	getline(file, line);

    while (getline(file, line)) {
            istringstream iss(line);
            string token;

            // Read the values of Date and Time columns
            getline(iss, token, ';'); // Date
            string date = token;
            getline(iss, token, ';'); // Time
            date += " " + token; // Combine Date and Time

            // Read the value of the Global_active_power column
            getline(iss, token, ';');
            float power;
            // In case of missing data (symbol "?") power gets 0
            istringstream(token) >> power;
            
            // Add the values to the vectors
            dates.push_back(date);
            values.push_back(power);
        }

    // Remeber to close the file
    file.close();

    // Create a bank of filters
    vector<float> temp_filters;
    vector<float> tmp = create_filter_trend_n_weeks(1, true);

    for(int i = 0; i < N_FILTERS; i++){
        temp_filters.insert(temp_filters.end(), tmp.begin(), tmp.end());
    }
    const vector<float> filters = temp_filters;

    cout << "Executing benchmark..." << endl;
    auto start_benchmark = chrono::high_resolution_clock::now();
    
    if(flt_type == SAD){
        const int block_size = 256;

        // Allocate CPU space for the result
        vector<float> SADs(N_FILTERS * SERIES_LENGTH, 0.0f);

        // Allocate GPU space for data values, results and filters
        float *d_values, *d_SADs, *d_filters;
        checkCudaErrors(hipMalloc((void**)&d_values, SERIES_LENGTH * sizeof(float)));
        checkCudaErrors(hipMalloc((void**)&d_SADs, N_FILTERS * SERIES_LENGTH * sizeof(float)));
        checkCudaErrors(hipMalloc((void**)&d_filters, filters.size() * sizeof(float)));

        // Copying data values and filters from CPU to GPU
        checkCudaErrors(hipMemcpy(d_values, values.data(), SERIES_LENGTH * sizeof(float), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_filters , filters.data(), filters.size() * sizeof(float), hipMemcpyHostToDevice));


        const int window_size = FILTER_LENGTH;
        const int n_blocks = (SERIES_LENGTH - window_size + (block_size - 1)) / block_size;
        calculate_SADs<<<n_blocks, block_size>>>(d_values, d_SADs, d_filters, window_size, SERIES_LENGTH);
        checkCudaErrors(hipGetLastError());

        // Wait for all kernels execution to finish
        checkCudaErrors(hipDeviceSynchronize());
        // Copy results from GPU to CPU
        checkCudaErrors(hipMemcpy(SADs.data(), d_SADs, N_FILTERS * SERIES_LENGTH * sizeof(float), hipMemcpyDeviceToHost));

        // Free all resources
        hipFree(d_filters);
        hipFree(d_values);
        hipFree(d_SADs);

        auto stop_benchmark = chrono::high_resolution_clock::now();
        auto duration_benchmark = chrono::duration_cast<chrono::milliseconds >(stop_benchmark - start_benchmark).count();
        
        cout << "Benchmark elapsed time: " << duration_benchmark << " ms" << endl;

        // Save data
        for(int filter_idx = 0; filter_idx < N_FILTERS; filter_idx++){
            string file_name = "../output_data/SAD"+ to_string(filter_idx) + "_filterlen" + to_string(FILTER_LENGTH) +".txt";
            ofstream output_file(file_name);
            if (output_file.is_open()) {
                for (int i = 0; i < SERIES_LENGTH; i++){
                    output_file << SADs[filter_idx * SERIES_LENGTH + i] << "\n";
                }
                output_file.close();
                cout << "Saved successfully: " << file_name << endl;
            } else {
                cerr << "Failed to open: " << file_name << endl;
            }
        }
    }
    else if(flt_type == ZMNCC){
        // Allocate CPU memory for the results
        vector<float> means(SERIES_LENGTH, 0.0f);
        vector<float> stds(SERIES_LENGTH, 0.0f);
        vector<float> zmnccs(N_FILTERS * SERIES_LENGTH, 0.0f);
        vector<float> filt_means(N_FILTERS, 0.0f);
        vector<float> filt_stds(N_FILTERS, 0.0f);

        // Allocate GPU memory
        float *d_values, *d_means, *d_stds, *d_zmnccs, *d_filters, *d_filt_means, *d_filt_stds;
        checkCudaErrors(hipMalloc((void**)&d_values, SERIES_LENGTH * sizeof(float)));
        checkCudaErrors(hipMalloc((void**)&d_means, SERIES_LENGTH * sizeof(float)));
        checkCudaErrors(hipMalloc((void**)&d_stds, SERIES_LENGTH * sizeof(float)));
        checkCudaErrors(hipMalloc((void**)&d_zmnccs, N_FILTERS * SERIES_LENGTH * sizeof(float)));
        checkCudaErrors(hipMalloc((void**)&d_filters, N_FILTERS * FILTER_LENGTH * sizeof(float)));
        checkCudaErrors(hipMalloc((void**)&d_filt_means, N_FILTERS * sizeof(float)));
        checkCudaErrors(hipMalloc((void**)&d_filt_stds, N_FILTERS * sizeof(float)));
        
        // Copying data from CPU to GPU
        checkCudaErrors(hipMemcpy(d_values, values.data(), SERIES_LENGTH * sizeof(float), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_filters, filters.data(), N_FILTERS * FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice));

        // Calculate mean and std of filters
        for (int filter_idx = 0; filter_idx < N_FILTERS; filter_idx++){
            const int window_size = FILTER_LENGTH;
            float filt_mean, filt_std;

            float sum = 0.0f;
            for (int i = 0; i <= window_size; ++i) {
                sum += filters[i + filter_idx * FILTER_LENGTH];
            }
            filt_mean = sum / window_size;

            float variance_summation = 0.0f;
            for (int i = 0; i < window_size; ++i) {
                variance_summation += pow(filters[i + filter_idx * FILTER_LENGTH] - filt_mean, 2);
            }
            filt_std = sqrt(variance_summation / (window_size - 1));
            
            // Save the mean and std of filter
            filt_means[filter_idx] = filt_mean;
            filt_stds[filter_idx] = filt_std;
        }
        cout << "Finished means and stds of filters" << endl;

        // Copying data from CPU to GPU
        checkCudaErrors(hipMemcpy(d_filt_means, filt_means.data(), N_FILTERS * sizeof(float), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_filt_stds, filt_stds.data(), N_FILTERS * sizeof(float), hipMemcpyHostToDevice));
        
        const int block_size = 256;
        const int window_size = FILTER_LENGTH;
        const int n_blocks = (SERIES_LENGTH - window_size + (block_size - 1)) / block_size;

        calculate_means_windowed<<<n_blocks, block_size>>>(d_values, d_means, window_size, SERIES_LENGTH);
        checkCudaErrors(hipGetLastError());
        // Wait for the kernel execution to finish to compute means
        checkCudaErrors(hipDeviceSynchronize());

        calculate_stds_zmnccs_windowed<<<n_blocks, block_size>>>(d_values, d_means, d_stds, d_zmnccs, d_filters, d_filt_means, d_filt_stds, window_size, SERIES_LENGTH);
        checkCudaErrors(hipGetLastError());
        // Wait for the kernel execution to finish compute stds and zmnccs
        checkCudaErrors(hipDeviceSynchronize());


        // Copy results from GPU to CPU
        // checkCudaErrors(hipMemcpy(means.data(), d_means, SERIES_LENGTH * sizeof(float), hipMemcpyDeviceToHost));
        // checkCudaErrors(hipMemcpy(stds.data(), d_stds, SERIES_LENGTH * sizeof(float), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(zmnccs.data(), d_zmnccs, N_FILTERS * SERIES_LENGTH * sizeof(float), hipMemcpyDeviceToHost));  

        // Free all resources
        hipFree(d_values);
        hipFree(d_means);
        hipFree(d_stds);
        hipFree(d_zmnccs);
        hipFree(d_filters);
        hipFree(d_filt_means);
        hipFree(d_filt_stds);

        auto stop_benchmark = chrono::high_resolution_clock::now();
        auto duration_benchmark = chrono::duration_cast<chrono::milliseconds >(stop_benchmark - start_benchmark).count();
        
        cout << "Benchmark elapsed time: " << duration_benchmark << " ms" << endl;

        // Save data
        for(int filter_idx = 0; filter_idx < N_FILTERS; filter_idx++){
            string file_name = "../output_data/zmncc"+ to_string(filter_idx) + "_filterlen" + to_string(FILTER_LENGTH) +".txt";
            ofstream output_file(file_name);
            if (output_file.is_open()) {
                for (int i = 0; i < SERIES_LENGTH; i++) {
                    output_file << zmnccs[filter_idx * SERIES_LENGTH + i] << "\n";
                }
                output_file.close();
                cout << "Saved successfully: " << file_name << endl;
            } else {
                cerr << "Failed to open: " << file_name << endl;
            }
        }
    }


    return 0;
}